#include "hip/hip_runtime.h"
// @file       demoKernel.cu
// @author     Charles Hatt <hatt@wisc.edu>
// @date       Apr-25-2015
// Please reference this work if you used it for your research!

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>



//These functions are defined in the demoKernel.cu file
hipError_t CUSetupTexture(hipArray* d_I, hipChannelFormatDesc chDesc);
hipError_t CUGetCurrentCost(float* h_curxy,float* h_curcost, float* d_curcost);

/////////Textures//////////////////
texture<float, 2, hipReadModeElementType> texIMG;


/////////Constant data//////////////////
__constant__ float cons_xy[2];


__global__ void kernelComputeCostFunction(float* d_cost)
{

    float x   = cons_xy[0];
    float y   = cons_xy[1];

    //Fetch the value of the image at coordinate x,y
    d_cost[0] = tex2D(texIMG,x-0.5,y-0.5);

}

hipError_t CUGetCurrentCost(float* h_curxy,float* h_curcost, float* d_curcost)
{
    hipError_t status;

    status = hipMemcpyToSymbol(HIP_SYMBOL(cons_xy), h_curxy, 2*sizeof(float));
    if(status != hipSuccess){
        printf("Error x y params to constant memory\n");
        return status;
    }

    //Call the kernel.  In this simple example, only 1 thread and 1 block are launched
    kernelComputeCostFunction<<<1,1>>>(d_curcost);
    status = hipGetLastError();
    if(status != hipSuccess){
        printf("Error running the CUDA kernel");
        return status;
    }

    status = hipMemcpy(h_curcost, d_curcost, 1*sizeof(float), hipMemcpyDeviceToHost);
    if (status != hipSuccess){
        printf("Copying cost back to the host failed\n");
        return status;
    }
    return status;
}


hipError_t CUSetupTexture(hipArray* d_I, hipChannelFormatDesc chDesc)
{
    hipError_t status;

    texIMG.addressMode[0] = hipAddressModeClamp;
    texIMG.addressMode[1] = hipAddressModeClamp;
    texIMG.filterMode     = hipFilterModeLinear;
    texIMG.normalized     = false;

    status = hipGetLastError();
    if(status != hipSuccess){
        printf("Error setting texture parameters\n");
        return status;
    }

    status = hipBindTextureToArray(texIMG, d_I, chDesc);
    if (status != hipSuccess){
        printf("Binding the texture failed\n");
        return status;
    }

    return status;
}


